#include "hip/hip_runtime.h"
/*
* gpu_conv_3.cu is a program to take matlab images and a 3x3 kernel and return a 2D convolution
*
*  
* we expect a format of [im_conv] = image_process [i1, kernel];
* AJN 3/18/19
*/

#include "mex.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#define PI 3.14159265358979323846
#define O_TILE_WIDTH 20								// variable to determine how many output tiles will be considered in a block
#define pix 17
# define BLOCK_WIDTH (O_TILE_WIDTH + (pix-1))		// block width needs to be output tiles + mask_width - 1 to ensure enough pixels are covered for calculation


/*****************************************************************************************************************************************************************************************/
/*
*															Device code
*
*  Multiple instances of the same void __global__ are created to deal with multiple size elements of kernels
*/
/*****************************************************************************************************************************************************************************************/


/*****************************************************************************************************************************************************************************************/
/*
COVOLUTION SECTION
*/
/*****************************************************************************************************************************************************************************************/

/*****COVOLVE********************************************************************************************************/

void __global__ convolve(double *d_iall,   // the gaussian is a separable filter and be treated as such
	double *d_gauss,	// makes these elements eligible for constant caching
	double *d_ifin,
	int irow,
	int icol,
	int gausw,				// width of guassian kernel
	int numi)
{
	// Declare variables
	__shared__ double d_i2[(BLOCK_WIDTH)][(BLOCK_WIDTH)];		// preallocate space for shared image
	double d_igauss[pix][pix];					// preallocate space for image mask

	// Coordinate building
	int tx = threadIdx.x;			// local x coord
	int ty = threadIdx.y;			// local y coord

	// location of output pixel being analyzed
	int row_output = blockIdx.y*O_TILE_WIDTH + ty;		// gives y coordinate as a function of tile width    **these lose meaning for (ty || tx) >= O_TILE_WIDTH and the same is true for **
	int col_output = blockIdx.x*O_TILE_WIDTH + tx;		// gives x coordinate as a function of tile width
	int imnum = blockIdx.z;

	// initialize location of apron		this forces the first pixel to take care of both the first output pixel, and loading the first input pixel
	// BLOCK_WIDTH is larger than O_TILE_WIDTH so there are more threads being used than output pixels being calculated
	int row_input = row_output - gausw / 2;	// EACH thread should load 1 input tile to the shared image as there are [BLOCK_WIDTH]x[BLOCK_WIDTH] threads in a block
	int col_input = col_output - gausw / 2;	// and BLOCK_WIDTH = O_TILE_WIDTH + MASK_WIDTH-1

	// Buffer data into block
	for (int grow = 0; grow < gausw; grow++){   // buffer gaussian kernel into block   **THIS HAS BEEN CHECKED AND VERIFIED TO WORK CORRECTLY **
		for (int gcol = 0; gcol < gausw; gcol++){
			d_igauss[grow][gcol] = d_gauss[grow + gcol*gausw];
		}
	}

	// build shared image into d_i2											// THIS HAS BEEN CHECKED TO BE LOADING CORRECTLY					
	// row/col_input represents the row/col of the input pixel being considered by 
	// thread [blockIdx.y*BLOCK_WIDTH+ty][blockIdx.x*BLOCK_WIDTH+tx]
	if ((row_input >= 0) && (row_input < irow) && (col_input >= 0) && (col_input < icol)){		// if statement checks the row/col indices to ensure they fall onto the input image
		d_i2[ty][tx] = d_iall[row_input + col_input*irow + imnum*irow*icol];										// if true, the value of the image is written to the shared array at location d_i2[ty][tx] and stored locally
	}																							// on the block
	else{
		d_i2[ty][tx] = 0.0*PI;																	// If row/col do not satisfy boundary condtions then assign a 0 to the value to build and apron of 
	}																							// of pixels that will not contribute to the calculation
	__syncthreads();
	// convolution calculation
	double d_res = 0.0*PI;		// initialize counting variable on thread register
	if (ty < O_TILE_WIDTH && tx < O_TILE_WIDTH) {										// check that the local thread should be apart of the calcualtion
		for (int rowcount = 0; rowcount < gausw; rowcount++){
			for (int colcount = 0; colcount < gausw; colcount++){
				d_res += d_i2[rowcount + ty][colcount + tx] * d_igauss[rowcount][colcount];
			}
		}
		if (row_output < irow && col_output < icol){
			d_ifin[row_output + col_output*irow + imnum*irow*icol] = d_res;			// assign to output variable  THIS SECTION WILL CORRECTLY WRITE TO d_ifin
		}
	}
}

/*****************************************************************************************************************************************************************************************/
/*****************************************************************************************************************************************************************************************/
/*
* Host code
*
*
*/
/*****************************************************************************************************************************************************************************************/
/*****************************************************************************************************************************************************************************************/


void mexFunction(int nlhs, mxArray *plhs[],
	int nrhs, mxArray const *prhs[])
{
	/* Declare all variables.*/
	double *iall;				// the pointer to the array of all images to be analyzed
	double *gauss_vec;
	double  *d_iall;			// Pointer to image array on gpu
	double *d_gauss;
	double *d_ifin;
	int irow;				// number of pixels in a row which should also be the number in a coloumn
	int icol;
	int numi;				// number of images imported
	const size_t *idims, *gaudims;
	hipDeviceReset();

    // Error Prevention Section
    if (nlhs != 1){
		printf("You must have 1 output variables [i_conv]\n");
		mexErrMsgTxt("See Error above!\n");
    }
    if (nrhs != 2){
        printf("You must have 2 input variables (i1, kernel)\n");
		mexErrMsgTxt("See Error above!\n");
    }

	// get pointer to input arguments
	iall = (double *)mxGetPr(prhs[0]);		// matlab linearizes in a coloumn major format which affects indexing (Writing MAtlab C/MEX Code - Research Gate)
	idims = mxGetDimensions(prhs[0]);	// get dimensions of image array
	irow = (int)idims[0];
	icol = (int)idims[1];
	numi = (int)idims[2];
	if (numi > 10000000 || numi < 1){
		numi = 1;
	}

	// get gauss dims
	gauss_vec = (double *)mxGetPr(prhs[1]);
	gaudims = mxGetDimensions(prhs[1]);
	int gaurow = (int)gaudims[0];
	int gaucol = (int)gaudims[1];

	// allocate memory on the gpu device
	hipError_t err1 = hipMalloc((void**)&d_iall, irow*icol*numi*sizeof(double));				// allocate image memory
	if (err1 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err1), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}

	hipError_t err2 = hipMalloc((void**)&d_gauss, gaurow*gaucol*sizeof(double));						// allocate gaussian memory
	if (err2 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err2), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}

	hipError_t err3 = hipMalloc((void**)&d_ifin, irow*icol*numi*sizeof(double));						// allocate completed image memory
	if (err3 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err3), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}
	

	// copy data from host to device
	hipError_t err9 = hipMemcpy(d_iall, iall, irow*icol*numi*sizeof(double), hipMemcpyHostToDevice);	// copy image data to gpu
	if (err9 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err9), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}

	hipError_t err10 = hipMemcpy(d_gauss, gauss_vec, gaurow*gaucol*sizeof(double), hipMemcpyHostToDevice);		// copy gauss data to gpu
	if (err10 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err10), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}

	/* Run GPU kernel*//* Run GPU kernel*/
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
	dim3 dimGrid((icol - 1) / O_TILE_WIDTH + 1, (irow - 1) / O_TILE_WIDTH + 1, numi);

	//							COVOLUTION SECTION IS COMMENTED OUT TO WORK ON EROSION
	// RUN THE CONVOLUTION WITH KERNEL 
	convolve << < dimGrid, dimBlock >> >(d_iall, d_gauss, d_ifin, irow, icol, gaurow, numi);
	plhs[0] = mxDuplicateArray(prhs[0]);
	double *ifin = (double *)mxGetPr(plhs[0]);
	
	//	printf("irow %d, icol %f, numi %f, line %d\n", numi, ifin[1], ifin[2], __LINE__);
	hipError_t err16 = hipMemcpy(ifin, d_ifin, irow*icol*numi*sizeof(double), hipMemcpyDeviceToHost);	// copy xf_all data
	if (err16 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err16), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}	
	hipFree(d_iall);
	hipFree(d_gauss);
	hipFree(d_ifin);
}