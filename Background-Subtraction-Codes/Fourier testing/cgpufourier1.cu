#include "hip/hip_runtime.h"
/*
 * cgpufourier is a program that will perform fourier 
 * transforms of a given variable on the gpu and subject them to a high pass filter 
 * this will return a "background subtracted" image
 * Calling in matlab will look like 
 * [iprod] = cgpufourier(i1, sp);
 * Written by Andrew Nelson 6/1/17
 * 
 * Need much better comments than this
 *
 *
 */
#include <mex.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


// gpu high pass filter
void __global__ highpass(hipfftComplex *data,
	int sp,
	int m,
	int n)
{
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if(index < n){
		/*double s = 0.0;
		double ms = 0;
		for (int i = 0; i<m; i++){
			ms = ms + powf(powf(data[index*m + i].x,2) + powf(data[index*m + i].y, 2),0.5);
		}
		ms = ms / (double)m;
		*/
		for (int i =0; i<sp*3; i++){
			data[index*m + i].x = powf(1 + exp((double)i-(double)sp),-1)*data[index*m+i].x;
			data[index*m + i].y = powf(1 + exp((double)i-(double)sp),-1)*data[index*m+i].y;
			data[index*m + (m-i-1)].y = powf(1 + exp((double)i-(double)sp),-1)*data[index*m + (m-i-1)].y;
			data[index*m + (m-i-1)].x = powf(1 + exp((double)i-(double)sp),-1)*data[index*m + (m-i-1)].x;
		}/*
		double as = 0;
		for (int i = 0; i<m; i++){
			as = as + powf(powf(data[index*m + i].x, 2) + powf(data[index*m + i].y, 2), 0.5);;
		}
		as = as/(double)m;
		s = ms/as;
		for (int i = 0; i<m; i++){
			data[index*m + i].x = s*data[index*m + i].x;
			data[index*m + i].y = s*data[index*m + i].y;
		}*/
	}
}

// gpu scale function
void __global__ scaleit(hipfftComplex *data,
	int top)
{
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if(index < top)
	{
		data[index].x = data[index].x/1000.0;
		data[index].y = data[index].y/1000.0;
	}
}

// convert data to handle complex
void pack_r2c(hipfftComplex  *output_float,
	double *input_re,
	int Ntot)
{
	int i;
	for (i = 0; i < Ntot; i++)
	{
		output_float[i].x = (float)input_re[i];
		output_float[i].y = 0.0;
	}
}

void pack_c2c(hipfftComplex  *output_float,
	double *input_re,
	double *input_im,
	int Ntot)
{
	int i;
	for (i = 0; i < Ntot; i++)
	{
		output_float[i].x = (float)input_re[i];
		output_float[i].y = (float)input_im[i];	}
}

void unpack_c2c(hipfftComplex  *input_float,
	double *output_re,
	double *output_im,
	int Ntot)
{
	int i;
	for (i = 0; i < Ntot; i++)
	{
		output_re[i] = (double)input_float[i].x;
		output_im[i] = (double)input_float[i].y;
	}
}

// main
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{

	// Variable Declaration
	double *yr,*yor, *yoi;
	int sp;
	sp = mxGetScalar(prhs[1]);
	hipfftComplex *data, *d_data, *y_out;

	
	printf("starting Fourier...\n");

		
	// Get memory size of signal
	const size_t *dims;
	dims = mxGetDimensions(prhs[0]);
	int m = (int)dims[0];
	int n = (int)dims[1];
	const int mem_size = m*n*sizeof(hipfftComplex);
	 
	
	// allocate space on host for data
	data = (hipfftComplex *)mxMalloc(mem_size);
	y_out = (hipfftComplex *)mxMalloc(mem_size);
	yr = (double *)mxGetPr(prhs[0]);
	// arrange the input to be complex data
	pack_r2c(data,yr,n*m);
	
	
	// allocate space on device for signal
	checkCudaErrors(hipMalloc((void**)&d_data, mem_size));	
	// Copy data over to device
	checkCudaErrors(hipMemcpy(d_data, data, mem_size, hipMemcpyHostToDevice));
	
	// at this point the memory is on the GPU ready to be manipulated
	hipfftHandle plan;
	checkCudaErrors(hipfftPlan1d(&plan, m, HIPFFT_C2C, n));
	checkCudaErrors(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD));
	
	// at this point the fourier transform is complete and sitting on the gpu
	highpass <<<((n-1)/1024 +1) ,1024>>>(d_data,sp,m,n);
	checkCudaErrors(hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD));
	//scaleit <<<((m*n-1)/1024 +1) ,1024>>>(d_data,m*n);
	// Copy data over from device
	
	checkCudaErrors(hipMemcpy(y_out,d_data,mem_size, hipMemcpyDeviceToHost));
	hipfftDestroy(plan);
	hipFree(d_data);
	// create complex double in matlab
	plhs[0] = mxCreateDoubleMatrix(m, n, mxCOMPLEX);


	yor = mxGetPr(plhs[0]);
	yoi = mxGetPi(plhs[0]);
	unpack_c2c(y_out, yor, yoi, n*m);
	mxFree(data);
}

