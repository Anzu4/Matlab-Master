#include "hip/hip_runtime.h"
/*
 * cgpufourier is a program that will perform fourier 
 * transforms of a given variable on the gpu and subject them to a high pass filter 
 * this will return a "background subtracted" image
 * Calling in matlab will look like 
 * [iprod] = cgpufourier(i1, sp);
 * Written by Andrew Nelson 6/1/17
 * 
 * Need much better comments than this
 *
 *
 */
#include <mex.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


// gpu high pass filter
void __global__ highpass(hipfftComplex *data,
	int sp,
	int m,
	int n)
{
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if(index < n){
		/*double s = 0.0;
		double ms = 0;
		for (int i = 0; i<m; i++){
			ms = ms + powf(powf(data[index*m + i].x,2) + powf(data[index*m + i].y, 2),0.5);
		}
		ms = ms / (double)m;
		*/
		for (int i =0; i<sp; i++){
			/*data[index*m + i].x = powf(1 + exp((double)i-(double)sp),-1)*data[index*m+i].x;
			data[index*m + i].y = powf(1 + exp((double)i-(double)sp),-1)*data[index*m+i].y;
			data[index*m + (m-i-1)].y = powf(1 + exp((double)i-(double)sp),-1)*data[index*m + (m-i-1)].y;
			data[index*m + (m-i-1)].x = powf(1 + exp((double)i-(double)sp),-1)*data[index*m + (m-i-1)].x;*/
			data[index*m + i].x = 0;
			data[index*m + i].y = 0;
			data[index*m + (m - i - 1)].y = 0;
			data[index*m + (m - i - 1)].x = 0;
		}
	}
}

void __global__ easypass(hipfftComplex *data,
	int sp,
	int m,
	int n)
{ 
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if(index < n*m){
		int b = index % m;
		if (b < sp || b >= m-sp ){
			if (b != 0){
				data[index].x = 0.0;
				data[index].y = 0.0;
			}
		}
	}
}
// gpu scale function
void __global__ scaleit(hipfftComplex *data,
	int top)
{
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if(index < top)
	{
		data[index].x = data[index].x/1000.0;
		data[index].y = data[index].y/1000.0;
	}
}

// convert data to handle complex
void pack_r2c(hipfftComplex  *output_float,
	float *input_re,
	int Ntot)
{
	
	int i;
	printf("Ntot is = %d\n", Ntot);
	for (i = 0; i < Ntot; i++)
	{
		
		output_float[i].x = input_re[i];
		output_float[i].y = 0.0;
	}
}

void pack_c2c(hipfftComplex  *output_float,
	double *input_re,
	double *input_im,
	int Ntot)
{
	int i;
	for (i = 0; i < Ntot; i++)
	{
		output_float[i].x = (float)input_re[i];
		output_float[i].y = (float)input_im[i];	}
}

void unpack_c2c(hipfftComplex  *input_float,
	double *output_re,
	double *output_im,
	int Ntot)
{
	int i;
	for (i = 0; i < Ntot; i++)
	{
		output_re[i] = (double)input_float[i].x;
		output_im[i] = (double)input_float[i].y;
	}
}

// main
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{

	// Variable Declaration
	double *yor, *yoi;
	float *yr;
	int sp;
	sp = mxGetScalar(prhs[1]);
	hipfftComplex *data, *d_data, *y_out;

	
	printf("starting Fourier...\n");

		
	// Get memory size of signal
	const size_t *dims;
	dims = mxGetDimensions(prhs[0]);
	int m = (int)dims[0];
	int n = (int)dims[1];
	const int mem_size = m*n*sizeof(hipfftComplex);
	 
	
	// allocate space on host for data
	data = (hipfftComplex *)mxMalloc(mem_size);
	y_out = (hipfftComplex *)mxMalloc(mem_size);
	yr = (float *)mxGetPr(prhs[0]);
	// arrange the input to be complex data
	printf("Before Packing n = %d and m = %d\n", n,m);
	pack_r2c(data,yr,n*m);
	printf("Data Allocating\n");	
	// allocate space on device for signal
	checkCudaErrors(hipMalloc((void**)&d_data, mem_size));	
	// Copy data over to device
	checkCudaErrors(hipMemcpy(d_data, data, mem_size, hipMemcpyHostToDevice));
	
	printf("Building Plan\n");
	// at this point the memory is on the GPU ready to be manipulated
	hipfftHandle plan;
	checkCudaErrors(hipfftPlan1d(&plan, m, HIPFFT_C2C, n));
	checkCudaErrors(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD));
	printf("Executing GPU\n");
	// at this point the fourier transform is complete and sitting on the gpu
	//highpass <<<((n-1)/1024 +1) ,1024>>>(d_data,sp,m,n);
	easypass << <(n*m - 1) / 1024 + 1, 1024 >> > (d_data, sp, m, n);
	checkCudaErrors(hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD));
	//scaleit <<<((m*n-1)/1024 +1) ,1024>>>(d_data,m*n);
	// Copy data over from device
	printf("Collecting from GPU\n");
	checkCudaErrors(hipMemcpy(y_out,d_data,mem_size, hipMemcpyDeviceToHost));
	hipfftDestroy(plan);
	hipFree(d_data);
	// create complex double in matlab
	plhs[0] = mxCreateDoubleMatrix(m, n, mxCOMPLEX);

	printf("Back to you Matlab\n");
	yor = mxGetPr(plhs[0]);
	yoi = mxGetPi(plhs[0]);
	unpack_c2c(y_out, yor, yoi, n*m);
	mxFree(data);
}

