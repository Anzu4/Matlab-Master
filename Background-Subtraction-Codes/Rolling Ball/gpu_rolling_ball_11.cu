#include "hip/hip_runtime.h"
/*
* image_process.cu is a program to take matlab images and process them through background subtraction, and segmentation for localization
*
*  v 0.2
Image convolution has been implemented successfully and checked with output from matlab. This can reliable convolve stacks of
*
V 0.1
we expect a format of [im_conv] = image_process [i1, i_gauss, i_ball];
*/

#include "mex.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#define PI 3.14159265358979323846
#define O_TILE_WIDTH 32								// variable to determine how many output tiles will be considered in a block
#define pix 11
# define BLOCK_WIDTH (O_TILE_WIDTH + (pix-1))		// block width needs to be output tiles + mask_width - 1 to ensure enough pixels are covered for calculation


/*****************************************************************************************************************************************************************************************/
/*
*															Device code
*
*  Multiple instances of the same void __global__ are created to deal with multiple size elements of kernels
*/
/*****************************************************************************************************************************************************************************************/


/*****************************************************************************************************************************************************************************************/
/*
COVOLUTION SECTION
*/
/*****************************************************************************************************************************************************************************************/

/*****COVOLVE 11********************************************************************************************************/

void __global__ convolve11(double *d_iall,   // the gaussian is a separable filter and be treated as such
	double *d_gauss,	// makes these elements eligible for constant caching
	double *d_ifin,
	int irow,
	int icol,
	int gausw,				// width of guassian kernel
	int numi)
{
	// Declare variables

	__shared__ double d_i2[(BLOCK_WIDTH)][(BLOCK_WIDTH)];		// preallocate space for shared image
	__shared__ double d_igauss[pix][pix];					// preallocate space for image mask

	// Coordinate building
	int tx = threadIdx.x;			// local x coord
	int ty = threadIdx.y;			// local y coord

	// location of output pixel being analyzed
	int row_output = blockIdx.y*O_TILE_WIDTH + ty;		// gives y coordinate as a function of tile width    **these lose meaning for (ty || tx) >= O_TILE_WIDTH and the same is true for **
	int col_output = blockIdx.x*O_TILE_WIDTH + tx;		// gives x coordinate as a function of tile width
	int imnum = blockIdx.z;

	// initialize location of apron		this forces the first pixel to take care of both the first output pixel, and loading the first input pixel
	// BLOCK_WIDTH is larger than O_TILE_WIDTH so there are more threads being used than output pixels being calculated
	int row_input = row_output - gausw / 2;	// EACH thread should load 1 input tile to the shared image as there are [BLOCK_WIDTH]x[BLOCK_WIDTH] threads in a block
	int col_input = col_output - gausw / 2;	// and BLOCK_WIDTH = O_TILE_WIDTH + MASK_WIDTH-1

	// Buffer data into block
	for (int grow = 0; grow < gausw; grow++){   // buffer gaussian kernel into block   **THIS HAS BEEN CHECKED AND VERIFIED TO WORK CORRECTLY **
		for (int gcol = 0; gcol < gausw; gcol++){
			d_igauss[grow][gcol] = d_gauss[grow + gcol*gausw];
		}
	}

	// build shared image into d_i2											// THIS HAS BEEN CHECKED TO BE LOADING CORRECTLY					
	// row/col_input represents the row/col of the input pixel being considered by 
	// thread [blockIdx.y*BLOCK_WIDTH+ty][blockIdx.x*BLOCK_WIDTH+tx]
	if ((row_input >= 0) && (row_input < irow) && (col_input >= 0) && (col_input < icol)){		// if statement checks the row/col indices to ensure they fall onto the input image
		d_i2[ty][tx] = d_iall[row_input + col_input*irow + imnum*irow*icol];										// if true, the value of the image is written to the shared array at location d_i2[ty][tx] and stored locally
	}																							// on the block
	else{
		d_i2[ty][tx] = 0.0*PI;																	// If row/col do not satisfy boundary condtions then assign a 0 to the value to build and apron of 
	}																							// of pixels that will not contribute to the calculation

	__syncthreads();																			// each thread uploads to a shared array later accessed by all threads, it is imperative to synch threads here

	// convolution calculation
	double d_res = 0.0*PI;		// initialize counting variable on thread register
	if (ty < O_TILE_WIDTH && tx < O_TILE_WIDTH) {										// check that the local thread should be apart of the calcualtion
		for (int rowcount = 0; rowcount < gausw; rowcount++){
			for (int colcount = 0; colcount < gausw; colcount++){
				d_res += d_i2[rowcount + ty][colcount + tx] * d_igauss[rowcount][colcount];
			}
		}
		if (row_output < irow && col_output < icol){


			d_ifin[row_output + col_output*irow + imnum*irow*icol] = d_res;			// assign to output variable  THIS SECTION WILL CORRECTLY WRITE TO d_ifin

		}

	}



}






/*****************************************************************************************************************************************************************************************/
/*****************************************************************************************************************************************************************************************/
/*
IMAGE EROSION SECTION
*/

/*****************************************************************************************************************************************************************************************/
/*****************************************************************************************************************************************************************************************/


/*****ERODE 11********************************************************************************************************/

void __global__ erode11(double *d_iall,   // the gaussian is a separable filter and be treated as such
	double *d_ball,	// makes these elements eligible for constant caching
	double *d_ifin,
	int irow,
	int icol,
	int ballw,				// width of guassian kernel
	int numi)
{
	// Declare variables

	__shared__ double d_i2[(BLOCK_WIDTH)][(BLOCK_WIDTH)];		// preallocate space for shared image
	__shared__ double d_iball[(11)][(11)];					// preallocate space for image mask

	// Coordinate building
	int tx = threadIdx.x;			// local x coord
	int ty = threadIdx.y;			// local y coord

	// location of output pixel being analyzed
	int row_output = blockIdx.y*O_TILE_WIDTH + ty;		// gives y coordinate as a function of tile width    **these lose meaning for (ty || tx) >= O_TILE_WIDTH and the same is true for **
	int col_output = blockIdx.x*O_TILE_WIDTH + tx;		// gives x coordinate as a function of tile width
	int imnum = blockIdx.z;
	if (imnum < numi){			// if there is an image to work on
		// initialize location of apron		this forces the first pixel to take care of both the first output pixel, and loading the first input pixel
		// BLOCK_WIDTH is larger than O_TILE_WIDTH so there are more threads being used than output pixels being calculated
		int row_input = row_output - ballw / 2;	// EACH thread should load 1 input tile to the shared image as there are [BLOCK_WIDTH]x[BLOCK_WIDTH] threads in a block
		int col_input = col_output - ballw / 2;	// and BLOCK_WIDTH = O_TILE_WIDTH + MASK_WIDTH-1

		// Buffer data into block
		for (int grow = 0; grow < ballw; grow++){   // buffer gaussian kernel into block   **THIS HAS BEEN CHECKED AND VERIFIED TO WORK CORRECTLY **
			for (int gcol = 0; gcol < ballw; gcol++){
				d_iball[grow][gcol] = d_ball[grow + gcol*ballw];
			}
		}

		// build shared image into d_i2											// THIS HAS BEEN CHECKED TO BE LOADING CORRECTLY					
		// row/col_input represents the row/col of the input pixel being considered by 
		// thread [blockIdx.y*BLOCK_WIDTH+ty][blockIdx.x*BLOCK_WIDTH+tx]
		if ((row_input >= 0) && (row_input < irow) && (col_input >= 0) && (col_input < icol)){		// if statement checks the row/col indices to ensure they fall onto the input image
			d_i2[ty][tx] = d_iall[row_input + col_input*irow + imnum*irow*icol];										// if true, the value of the image is written to the shared array at location d_i2[ty][tx] and stored locally
		}																							// on the block
		else{
			d_i2[ty][tx] = 0;																	// If row/col do not satisfy boundary condtions then assign a 0 to the value to build and apron of 
		}																							// of pixels that will not contribute to the calculation

		__syncthreads();																			// each thread uploads to a shared array later accessed by all threads, it is imperative to synch threads here

		// convolution calculation
		double d_res_min = 1000.0*PI;		// initialize counting variable on thread register
		double d_res;
		if (ty < O_TILE_WIDTH && tx < O_TILE_WIDTH) {										// check that the local thread should be apart of the calcualtion
			for (int rowcount = 0; rowcount < ballw; rowcount++){
				for (int colcount = 0; colcount < ballw; colcount++){
					d_res = d_i2[rowcount + ty][colcount + tx] - d_iball[rowcount][colcount];		// subtract height of ball from pixel height at each location and check for lowest value
					if (d_res < d_res_min){
						d_res_min = d_res;		// if found value is lower than housed value assign lowest value to lowest found
					}
				}	// end col loop
			} // end row loop
			if (row_output < irow && col_output < icol){


				d_ifin[row_output + col_output*irow + imnum*irow*icol] = d_res_min;			// assign to output variable  THIS SECTION WILL CORRECTLY WRITE TO d_ifin

			}// end if output pixel exists inside image

		}  // end if ty and tx are withing output tile
	} // end if picture exists


}		// end erosion






/*****************************************************************************************************************************************************************************************/
/*****************************************************************************************************************************************************************************************/
/*
IMAGE DIALATION SECTION
*/
/*
*			Unsure how to match MATLAB here  - 5/11/15 AJN
*
*
/*****************************************************************************************************************************************************************************************/
/*****************************************************************************************************************************************************************************************/




/*****DIALATE 11********************************************************************************************************/

void __global__ dilate11(double *d_iall,   // the gaussian is a separable filter and be treated as such
	double *d_ball,	// makes these elements eligible for constant caching
	double *d_ifin,
	int irow,
	int icol,
	int ballw,				// width of guassian kernel
	int numi)
{
	// Declare variables

	__shared__ double d_i2[(BLOCK_WIDTH)][(BLOCK_WIDTH)];		// preallocate space for shared image
	__shared__ double d_iball[(11)][(11)];					// preallocate space for image mask

	// Coordinate building
	int tx = threadIdx.x;			// local x coord
	int ty = threadIdx.y;			// local y coord

	// location of output pixel being analyzed
	int row_output = blockIdx.y*O_TILE_WIDTH + ty;		// gives y coordinate as a function of tile width    **these lose meaning for (ty || tx) >= O_TILE_WIDTH and the same is true for **
	int col_output = blockIdx.x*O_TILE_WIDTH + tx;		// gives x coordinate as a function of tile width
	int imnum = blockIdx.z;
	if (imnum < numi){			// if there is an image to work on
		// initialize location of apron		this forces the first pixel to take care of both the first output pixel, and loading the first input pixel
		// BLOCK_WIDTH is larger than O_TILE_WIDTH so there are more threads being used than output pixels being calculated
		int row_input = row_output - ballw / 2;	// EACH thread should load 1 input tile to the shared image as there are [BLOCK_WIDTH]x[BLOCK_WIDTH] threads in a block
		int col_input = col_output - ballw / 2;	// and BLOCK_WIDTH = O_TILE_WIDTH + MASK_WIDTH-1

		// Buffer data into block
		for (int grow = 0; grow < ballw; grow++){   // buffer gaussian kernel into block   **THIS HAS BEEN CHECKED AND VERIFIED TO WORK CORRECTLY **
			for (int gcol = 0; gcol < ballw; gcol++){
				d_iball[grow][gcol] = d_ball[grow + gcol*ballw];
			}
		}

		// build shared image into d_i2											// THIS HAS BEEN CHECKED TO BE LOADING CORRECTLY					
		// row/col_input represents the row/col of the input pixel being considered by 
		// thread [blockIdx.y*BLOCK_WIDTH+ty][blockIdx.x*BLOCK_WIDTH+tx]
		if ((row_input >= 0) && (row_input < irow) && (col_input >= 0) && (col_input < icol)){		// if statement checks the row/col indices to ensure they fall onto the input image
			d_i2[ty][tx] = d_iall[row_input + col_input*irow + imnum*irow*icol];										// if true, the value of the image is written to the shared array at location d_i2[ty][tx] and stored locally
		}																							// on the block
		else{
			d_i2[ty][tx] = 0;																	// If row/col do not satisfy boundary condtions then assign a 0 to the value to build and apron of 
		}																							// of pixels that will not contribute to the calculation

		__syncthreads();																			// each thread uploads to a shared array later accessed by all threads, it is imperative to synch threads here

		// convolution calculation
		double d_res_max = -100.0;		// initialize counting variable on thread register
		double d_res;
		if (ty < O_TILE_WIDTH && tx < O_TILE_WIDTH) {										// check that the local thread should be apart of the calcualtion
			for (int rowcount = 0; rowcount < ballw; rowcount++){
				for (int colcount = 0; colcount < ballw; colcount++){
					d_res = d_i2[rowcount + ty][colcount + tx] + d_iball[rowcount][colcount];		// add height of ball from pixel height at each location and check for maximm value
					if (d_res == d_iball[rowcount][colcount]){d_res =0;}
					if (d_res > d_res_max){
						d_res_max = d_res;		// if found value is lower than housed value assign lowest value to lowest found
					}
				}	// end col loop
			} // end row loop
			if (row_output < irow && col_output < icol){


				d_ifin[row_output + col_output*irow + imnum*irow*icol] = d_res_max;			// assign to output variable  THIS SECTION WILL CORRECTLY WRITE TO d_ifin

			}// end if output pixel exists inside image

		}  // end if ty and tx are withing output tile
	} // end if picture exists


}		// end dilation



/*****************************************************************************************************************************************************************************************/
/*****************************************************************************************************************************************************************************************/
/*

BACKGROUND SUBTRACTION STEP

*/
/*****************************************************************************************************************************************************************************************/
/*****************************************************************************************************************************************************************************************/



void __global__ bkgsub(double *d_iall,
	double *d_ibkg,
	int	numel)
{
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < numel){
		d_iall[index] = d_iall[index] - d_ibkg[index];
		if (d_iall[index] < 0.0){
			d_iall[index] = 0.0;
		}
	}
}


/*****************************************************************************************************************************************************************************************/
/*****************************************************************************************************************************************************************************************/
/*
* Host code
*
*
*/
/*****************************************************************************************************************************************************************************************/
/*****************************************************************************************************************************************************************************************/


void mexFunction(int nlhs, mxArray *plhs[],
	int nrhs, mxArray const *prhs[])
{
	/* Declare all variables.*/
	double *iall;				// the pointer to the array of all images to be analyzed
	double *gauss_vec;
	double *ball_vec;
	double  *d_iall;			// Pointer to image array on gpu
	double *d_gauss;
	double *d_ball;
	double *d_ifin, *d_ifin2;
	int irow;				// number of pixels in a row which should also be the number in a coloumn
	int icol;
	int numi;				// number of images imported
	const size_t *idims, *gaudims, *balldims;
	hipDeviceReset();

	
	// get pointer to input arguments
	iall = (double *)mxGetPr(prhs[0]);		// matlab linearizes in a coloumn major format which affects indexing (Writing MAtlab C/MEX Code - Research Gate)
	idims = mxGetDimensions(prhs[0]);	// get dimensions of image array
	icol = (int)idims[1];
	irow = (int)idims[0];
	numi = (int)idims[2];
	if (numi > 10000000 || numi < 1){
		numi = 1;
	}

	// get gauss dims
	gauss_vec = (double *)mxGetPr(prhs[1]);
	gaudims = mxGetDimensions(prhs[1]);
	int gaurow = (int)gaudims[0];
	int gaucol = (int)gaudims[1];

	// get ball dims
	ball_vec = (double *)mxGetPr(prhs[2]);
	balldims = mxGetDimensions(prhs[1]);
	int balrow = (int)gaudims[0];
	int balcol = (int)gaudims[1];




	if (nlhs != 2){
		printf("You must have 1 output variables [i_erode]\n");
		mexErrMsgTxt("See Error above!\n");
	}
	// allocate memory on the gpu device



	hipError_t err1 = hipMalloc((void**)&d_iall, irow*icol*numi*sizeof(double));				// allocate image memory
	if (err1 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err1), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}

	hipError_t err2 = hipMalloc((void**)&d_gauss, gaurow*gaucol*sizeof(double));						// allocate gaussian memory
	if (err2 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err2), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}

	hipError_t err3 = hipMalloc((void**)&d_ifin, irow*icol*numi*sizeof(double));						// allocate completed image memory
	if (err3 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err3), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}
	
	hipError_t err4 = hipMalloc((void**)&d_ball, balrow*balcol*sizeof(double));						// allocate completed image memory
	if (err4 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err4), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}

	// copy data from host to device
	hipError_t err9 = hipMemcpy(d_iall, iall, irow*icol*numi*sizeof(double), hipMemcpyHostToDevice);	// copy image data to gpu
	if (err9 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err9), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}

	hipError_t err10 = hipMemcpy(d_gauss, gauss_vec, gaurow*gaucol*sizeof(double), hipMemcpyHostToDevice);		// copy gauss data to gpu
	if (err10 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err10), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}


	hipError_t err1a = hipMemcpy(d_ball, ball_vec, balrow*balcol*sizeof(double), hipMemcpyHostToDevice);		// copy gauss data to gpu
	if (err1a != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err1a), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}
	/* Run GPU kernel*/
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
	dim3 dimGrid((icol - 1) / O_TILE_WIDTH + 1, (irow - 1) / O_TILE_WIDTH + 1, numi);

	//							COVOLUTION SECTION IS COMMENTED OUT TO WORK ON EROSION
	// RUN THE CONVOLUTION WITH KERNEL 

	convolve11 << < dimGrid, dimBlock >> >(d_iall, d_gauss, d_ifin, irow, icol, gaurow, numi);



	erode11 << <dimGrid, dimBlock >> > (d_ifin, d_ball, d_iall, irow, icol, balrow, numi);
	dilate11 << <dimGrid, dimBlock >> > (d_iall, d_ball, d_ifin, irow, icol, balrow, numi);
	// copy data from host to device
	hipError_t err89 = hipMemcpy(d_iall, iall, irow*icol*numi*sizeof(double), hipMemcpyHostToDevice);	// copy image data to gpu
	if (err89 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err89), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}
	int numel = irow*icol*numi;	// number of pixels in the entire image
	bkgsub<< <(numel-1)/1000+1, 1000 >> > (d_iall, d_ifin, numel);			// routiune vecor subtraction on GPU!!!!!













	/*		 copy data back to mxarray pointers for output
	*
	*
	*		Duplicate the input array of equal size to the output array
	*		Send the pointer to a variable
	*		copy data to place pointer points to, which is output
	*/
	


	plhs[0] = mxDuplicateArray(prhs[0]);
	double *ifin = (double *)mxGetPr(plhs[0]);
	plhs[1] = mxDuplicateArray(prhs[0]);
	double *ibkn = (double *)mxGetPr(plhs[1]);
	//	printf("irow %d, icol %f, numi %f, line %d\n", numi, ifin[1], ifin[2], __LINE__);
	hipError_t err16 = hipMemcpy(ifin, d_iall, irow*icol*numi*sizeof(double), hipMemcpyDeviceToHost);	// copy xf_all data
	if (err16 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err16), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}
	hipError_t err17 = hipMemcpy(ibkn, d_ifin, irow*icol*numi*sizeof(double), hipMemcpyDeviceToHost);	// copy xf_all data
	if (err16 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err16), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}
	//	printf("irow %f, icol %f, numi %f, line %d\n", ifin[0], ifin[1], ifin[2], __LINE__);

	// hipDeviceReset();
	 
	
	hipFree(d_iall);
	hipFree(d_gauss);
	hipFree(d_ifin);
	//hipFree(d_ifin2);
	hipFree(d_ball);
}